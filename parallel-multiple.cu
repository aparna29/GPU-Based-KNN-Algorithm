#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
__global__ void kernel_distance(double* dfeature_arr,int d_numfeatures,int d_querys, int d_querye, double* d_dist, double* d_label,int k,int num_blocks)
{
	int id = blockIdx.x*blockDim.x +  threadIdx.x;
	int tid = threadIdx.x;
	
	int i;
	int query_size = d_querye - d_querys;
	__shared__ double queryobj[20][15]; // 20-> querysize 15 -> num_features
	int start_q = (d_querys + tid -1)*d_numfeatures;
	if(tid<query_size)
	{
		for(i =0 ;i<d_numfeatures; i++)
		{
			queryobj[tid][i] = dfeature_arr[start_q];
			start_q++;
		}
	}
	__syncthreads();
	
	__shared__ double d_arr[258][20]; // 20 -> querysize
	__shared__ int check[258];
	double d1;
	check[tid] = 1;
	int rank,j;
	int start = id*d_numfeatures;
	int end = start + d_numfeatures-1;
	for(j = 0;j<query_size;j++)
	{
		double dis = 0;
		start_q = 0;
		for(i = start;i<end;i++)
		{
			d1 = dfeature_arr[i];
			dis += (d1 - queryobj[j][start_q])*(d1 - queryobj[j][start_q]);
			start_q++;
		}
		d_arr[tid][j] = dis;
	}
	//printf("\n Label = %lf",dfeature_arr[end]);
	__syncthreads();
	
	for(j=0;j<query_size;j++)
	{
		rank = 0;
		for(i=0;i<256;i++)
		{	
			//if(check[i]==1 && dis>d_arr[i]&& i!=tid)
			//	rank++;
			if(check[i]==1 &&i!=tid )
			{
				if(d_arr[tid][j]>d_arr[i][j])
					rank++;
				if(d_arr[tid][j]==d_arr[i][j]&&tid>i)
					rank++;
			}
		
		}
		if(rank<k)
		{
			d_dist[j*(k*num_blocks)+ blockIdx.x*k + rank] = d_arr[tid][j];
			d_label[j*(k*num_blocks)+ blockIdx.x*k + rank] = dfeature_arr[end];
			//printf("\nQuery_ object = %d BlockID = %d  Rank = %d  dist = %lf  label = %lf",j,blockIdx.x,rank,d_arr[tid][j],dfeature_arr[end]);
		}
	 }

}
__global__ void kernel_knn(double *dknn,int num_blocks, double* d_dist, double *d_label, int k)
{
    int tid = threadIdx.x;
    int start[100]; // 100 -> num_blocks
    double knn[20],mi; // 20 ->k
    int i,kid = 0;
    int j,ind;
    for(i=0;i<num_blocks;i++)
    {
		start[i] = tid*num_blocks*k +  k*i;
    }

    for(i=0;i<k;i++)
    {
		mi = 1000;
		for(j=0;j<num_blocks;j++)
		{
			if(mi>d_dist[start[j]])
			{
				mi = d_dist[start[j]];
				ind = j;
			}
		}
        //start[j]++;
	//printf("\n Distance = %lf label = %lf",mi,h_label[start[ind]]);
		knn[kid] = d_label[start[ind]];
		kid++;
		start[ind]++;
    }
    double sum=0.0;
    for(i=0;i<k;i++)
    {
		//printf("\n result = %lf",h_label[i]);
		sum+=knn[i];
    }
	sum = sum/(double)k;
	dknn[tid] = sum;
}
const char* getfield(char* line, int num)
{
    const char* tok;
    for (tok = strtok(line, ";");
            tok && *tok;
            tok = strtok(NULL, ";\n"))
    {
        if (!--num)
            return tok;
    }
    return NULL;
}
main()
{
    FILE* stream = fopen("winequality-red.csv", "r");

    char line[1024];
    int cnt = 1,num_features,itr =-1,i,index = 0;
    double *feature_arr;
    feature_arr = (double *)malloc(sizeof(double)*250000);
    while (fgets(line, 1024, stream)!=NULL)
    {
        char* tmp = strdup(line);
        if(itr==-1)
        {
        	while(getfield(tmp,cnt)!=NULL)
        	{	
        		cnt++;
        		tmp = strdup(line);
        	}
        	num_features = cnt -1;
        	printf("\n Number of features = %d",num_features);
        	itr++;
        }
        else
        {
		for(i=1;i<=num_features;i++)
        	{
        		//printf("feaure cnt = %d",i);
			feature_arr[index] = atof(getfield(tmp,i));
			index++;
        		tmp = strdup(line);
        	}
        	itr++;
        }
        free(tmp);
    }
    fclose(stream);
    printf("\n Reading done");
    double *dfeature_arr, *d_dist,*d_label;

    hipEvent_t st, stop;
    hipEventCreate(&st);
    hipEventCreate(&stop);
    hipMalloc((void **)&dfeature_arr,itr*num_features*sizeof(double));
    
    hipMemcpy(dfeature_arr,feature_arr,itr*num_features*sizeof(double),hipMemcpyHostToDevice);
    
    int train = 0.8*(float)itr;
    int j;
    printf("\n %d",train);
    for(j=1;j<=15;j++)
    {
    int querys = train + 10;
    int querye = train + 10 + j;
    int query_size = querye -querys;
    int k =5;
    //printf("\nEnter value of k - ");
    //scanf("%d",&k);
    double h_dist[train][query_size];
    double h_label[train][query_size];
    hipMalloc((void **)&d_dist,train*query_size*sizeof(double));
    hipMalloc((void **)&d_label,train*query_size*sizeof(double));
        
    int num_threads =256;
    int num_blocks = ceil((float)train/num_threads);
    
    hipEventRecord(st);
    kernel_distance<<<num_blocks,num_threads>>>(dfeature_arr,num_features,querys,querye,d_dist,d_label,k,num_blocks);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, st, stop);


    hipMemcpy(h_dist,d_dist,sizeof(double)*train*query_size,hipMemcpyDeviceToHost);
    hipMemcpy(h_label,d_label,sizeof(double)*train*query_size,hipMemcpyDeviceToHost);
    
   /* for(i=0;i<k*num_blocks;i++)
    {
	printf("\n Distance  = %lf  Label = %lf ",h_dist[i],h_label[i]);
    }*/

    double *knn;
    knn = (double *)malloc(sizeof(double)*query_size);
    
    double *dknn;
    hipMalloc((void **)&dknn,query_size*sizeof(double));

    double *d_dist2,*d_label2;
    hipMalloc((void **)&d_dist2,train*query_size*sizeof(double));
    hipMalloc((void **)&d_label2,train*query_size*sizeof(double));

    hipMemcpy(d_dist2,h_dist,sizeof(double)*train*query_size,hipMemcpyHostToDevice);
    hipMemcpy(d_label2,h_label,sizeof(double)*train*query_size,hipMemcpyHostToDevice);

    hipEvent_t st1, stop1;
    hipEventCreate(&st1);
    hipEventCreate(&stop1);

    hipEventRecord(st1);

    kernel_knn<<<1,query_size>>>(dknn, num_blocks, d_dist2, d_label2, k);
    hipEventRecord(stop1);


    hipEventSynchronize(stop1);
    float millisecond = 0;
    hipEventElapsedTime(&millisecond, st1, stop1);


    hipMemcpy(knn,dknn,sizeof(double)*query_size,hipMemcpyDeviceToHost);
    
   
    //for(i =0; i<query_size; i++)
    //printf("\nLabel for query %d = %lf",i,knn[i]);
    
    printf("\n%d\t%lf",j,(double)(milliseconds+ millisecond));
    }
}
