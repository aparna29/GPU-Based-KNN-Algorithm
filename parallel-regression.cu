#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
__global__ void kernel_distance(double* dfeature_arr,int d_numfeatures,int d_query,double* d_dist, double* d_label,int k)
{
	int id = blockIdx.x*blockDim.x +  threadIdx.x;
	int tid = threadIdx.x;
	__shared__ double d_arr[258];
	__shared__ int check[258];
	__shared__ double query_obj[50]; // 50 -> num_features
	double d1, d2;
	check[tid] = 1;
	int rank,j;
	int start = id*d_numfeatures;
	int end = start + d_numfeatures-1;
	int i;
	double dis = 0;
	int start_q = (d_query-1)*d_numfeatures;

	if(tid<d_numfeatures)
	query_obj[tid] = dfeature_arr[start_q + tid];
	__syncthreads();

	for(i = start, j= 0;i<end;i++, j++)
	{
		d1 = dfeature_arr[i];
		d2 = query_obj[j];
		dis += (d1 - d2)*(d1 - d2);
		//start_q++;
	}
	//d_dist[id] = dis;
	d_arr[tid] = dis;
	//printf("\n Label = %lf",dfeature_arr[end]);
	__syncthreads();
	rank = 0;
	for(i=0;i<256;i++)
	{
		//if(check[i]==1 && dis>d_arr[i]&& i!=tid)
		//	rank++;
		if(check[i]==1 &&i!=tid )
		{
			if(dis>d_arr[i])
				rank++;
			if(dis==d_arr[i]&&tid>i)
				rank++;
		}
		
	}

	if(rank<k)
	{
		d_dist[blockIdx.x*k+rank] = dis;
		d_label[blockIdx.x*k + rank] = dfeature_arr[end];
		//printf("\n BlockID = %d  Rank = %d  dist = %lf  label = %lf",blockIdx.x,rank,dis,d_label[blockIdx.x*k + rank]);
	}
}
const char* getfield(char* line, int num)
{
    const char* tok;
    for (tok = strtok(line, ";");
            tok && *tok;
            tok = strtok(NULL, ";\n"))
    {
        if (!--num)
            return tok;
    }
    return NULL;
}
main()
{
    FILE* stream = fopen("winequality-red.csv", "r");

    char line[1024];
    int cnt = 1,num_features,itr =-1,i,index = 0;
    double *feature_arr;
    feature_arr = (double *)malloc(sizeof(double)*250000);
    while (fgets(line, 1024, stream)!=NULL)
    {
        char* tmp = strdup(line);
        if(itr==-1)
        {
        	while(getfield(tmp,cnt)!=NULL)
        	{	
        		cnt++;
        		tmp = strdup(line);
        	}
        	num_features = cnt -1;
        	printf("\n Number of features = %d",num_features);
        	itr++;
        }
        else
        {
		for(i=1;i<=num_features;i++)
        	{
        		//printf("feaure cnt = %d",i);
			feature_arr[index] = atof(getfield(tmp,i));
			index++;
        		tmp = strdup(line);
        	}
        	itr++;
        }
        free(tmp);
    }
    fclose(stream);
    printf("\n Reading done");
    double *dfeature_arr, *d_dist,*d_label;

    hipEvent_t st, stop;
    hipEventCreate(&st);
    hipEventCreate(&stop);
    hipMalloc((void **)&dfeature_arr,itr*num_features*sizeof(double));
    
    hipMemcpy(dfeature_arr,feature_arr,itr*num_features*sizeof(double),hipMemcpyHostToDevice);
    
    int train = 0.8*(float)itr;
    printf("\n %d",train);
    int query = train + 10;
    int k ;
    //printf("\nEnter value of k - ");
    //scanf("%d",&k);
    //for(k=3;k<=20;k++){
    double h_dist[train];
    double h_label[train];
    hipMalloc((void **)&d_dist,train*sizeof(double));
    hipMalloc((void **)&d_label,train*sizeof(double));
        
    int num_threads =256;
    int num_blocks = ceil((float)train/num_threads);
     for(k=3;k<=20;k++){
    hipEventRecord(st);
    kernel_distance<<<num_blocks,num_threads>>>(dfeature_arr,num_features,query,d_dist,d_label,k);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, st, stop);


    hipMemcpy(h_dist,d_dist,sizeof(double)*train,hipMemcpyDeviceToHost);
    hipMemcpy(h_label,d_label,sizeof(double)*train,hipMemcpyDeviceToHost);
    
    int start[num_blocks];
    double knn[k],mi;
    int kid = 0;
    int j,ind;
    /*for(i=0;i<k*num_blocks;i++)
    {
	printf("\n %lf   %lf",h_label[i],h_dist[i]);
    }*/
    clock_t s, e;
    s = clock();
    for(i=0;i<num_blocks;i++)
    {
	start[i] = k*i;
	//printf("\n i = %d, start = %d",i,start[i]);
    }

    for(i=0;i<k;i++)
    {
	mi = 1000;
	for(j=0;j<num_blocks;j++)
	{
		if(mi>h_dist[start[j]])
		{
			mi = h_dist[start[j]];
			ind = j;
		}
	}
        //start[j]++;
	//printf("\n Distance = %lf label = %lf",mi,h_label[start[ind]]);
	knn[kid] = h_label[start[ind]];
	kid++;
	start[ind]++;
    }

    double sum=0.0;
    for(i=0;i<k;i++)
    {
		//printf("\n result = %lf",h_label[i]);
		sum+=knn[i];
    }
    //printf("\n Sum = %lf",sum);
    sum = sum/(double)k;
    e = clock();
    //printf("\nLabel = %lf",sum);
    
    printf("\n %d\t%lf",k,((double) (e - s))* 1000.0 / CLOCKS_PER_SEC +  (double)milliseconds);}
}
